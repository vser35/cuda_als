#include "hip/hip_runtime.h"
#include <memory>


#include "als_model.h"
#include "hip/hip_runtime.h"
#include "cuda_common.h"
#include "logger.h"

__global__
void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
	// left matrix is vt (i.e. columns from VT for items rated by current user)
	// top matrix is v (i.e. vt transposed)

	/* start = current row pointer
	 * end = next row pointer
	 * nr_of_items = end - start
	 * top_col = out_col = threadIdx.x
	 * for(left_row = out_row = 0; left_row < f; ++left_row ++ out_row)
	 *     temp = 0
	 *     for(curr_item_nr = 0; curr_item_nr < nr_of_items; ++curr_item)
	 *         left_col = top_row = curr_item_nr
	 *         // VT is column-major:
	 *         //     for left matrix, move to next row <=> add 1 to idx
	 *         //     for left matrix, move to next col <=> add f to idx
	 *         //     for top matrix, move to next row <=> add f to idx
	 *         //     for top matrix, move to next col <=> add 1 to idx
	 *
	 *         left_row_offset = left_row
	 *         left_col_offset = csr_col_idxs[start + left_col] * f
	 *
	 *         top_row_offset = csr_col_idxs[start + top_row] * f
	 *         top_col_offset = top_row
	 *
	 *         temp += VT[left_row_offset + left_col_offset] * VT[top_row_offset + top_col_offset]
	 *     vtvs[blockIdx.x * f * f + left_row + ] = temp
	 */

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int top_col = threadIdx.x;
		int out_col = top_col;

		int left_row = 0;
		int out_row = left_row;

		while(left_row < f) {
			float out = 0;

			for(int item_nr = 0; item_nr < items_cnt; ++ item_nr) {
				int left_col = item_nr;
				int top_row = left_col;

				// VT is column-major:
				//     for left matrix, move to next row <=> add 1 to idx
				//     for left matrix, move to next col <=> add f to idx
				//     for top matrix, move to next row <=> add f to idx
				//     for top matrix, move to next col <=> add 1 to idx

				int left_row_offset = left_row;
				int left_col_offset = csr_col_idxs[start + left_col] * f;

				int top_row_offset = csr_col_idxs[start + top_row] * f;
				int top_col_offset = top_col;

				out += VT[left_row_offset + left_col_offset] * VT[top_row_offset + top_col_offset];
			}

			// regularization
			if(left_row == top_col) {
				out += items_cnt * lambda;
			}

			vtvs[user_idx * f * f + out_row + out_col * f] = out;

			++left_row;
			++out_row;
		}
	}
}

__global__
void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
	extern __shared__ float smem [];

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int smem_iters = (items_cnt - 1) / smem_col_cnt + 1;


		int top_col = threadIdx.x;
		int out_col = top_col;

		for(int smem_iter = 0; smem_iter < smem_iters; ++smem_iter) {
			int smem_items = smem_col_cnt * (smem_iter + 1) < items_cnt ? smem_col_cnt : items_cnt - smem_col_cnt * smem_iter;

			// Each thread reads smem_col_cnt columns - all threads are busy
			for(int smem_col = 0; smem_col < smem_items; ++smem_col) {
				smem[f * smem_col + threadIdx.x] = VT[f * csr_col_idxs[start + smem_iter * smem_col_cnt + smem_col] + threadIdx.x];
			}

			__syncthreads();

			// actual work

			int left_row = 0;
			int out_row = left_row;

			while(left_row < f) {
				float out = 0;

				for(int item_nr = 0; item_nr < smem_items; ++item_nr) {
					// VT is column-major:
					//     for left matrix, move to next row <=> add 1 to idx
					//     for left matrix, move to next col <=> add f to idx
					//     for top matrix, move to next row <=> add f to idx
					//     for top matrix, move to next col <=> add 1 to idx

					int left_col = item_nr;
					int top_row = left_col;

					int left_row_offset = left_row;
					int left_col_offset = left_col * f;

					int top_row_offset = top_row * f;
					int top_col_offset = top_col;

					out += smem[left_row_offset + left_col_offset] * smem[top_row_offset + top_col_offset];
				}

				vtvs[user_idx * f * f + out_row + out_col * f] += out;

				++left_row;
				++out_row;
			}
		}

		// regularization
		vtvs[user_idx * f * f + out_col + out_col * f] += items_cnt * lambda;
	}
}

__global__
void calculate_vtvs_smem_row_major_no_calc(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
	extern __shared__ float smem [];

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int smem_iters = (items_cnt - 1) / smem_col_cnt + 1;


		int top_col = threadIdx.x;
		int out_col = top_col;

		for(int smem_iter = 0; smem_iter < smem_iters; ++smem_iter) {
			int smem_items = smem_col_cnt * (smem_iter + 1) < items_cnt ? smem_col_cnt : items_cnt - smem_col_cnt * smem_iter;

			// Each thread reads smem_col_cnt columns - all threads are busy
			for(int smem_col = 0; smem_col < smem_items; ++smem_col) {
				smem[f * smem_col + threadIdx.x] = VT[f * csr_col_idxs[start + smem_iter * smem_col_cnt + smem_col] + threadIdx.x];
			}

			__syncthreads();

			// no actual work - just measuring smem loading time
		}
	}
}

__global__
void calculate_vtvs_smem_col_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
	extern __shared__ float smem [];

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int smem_iters = (items_cnt - 1) / smem_col_cnt + 1;


		int top_col = threadIdx.x;
		int out_col = top_col;

		for(int smem_iter = 0; smem_iter < smem_iters; ++smem_iter) {
			int smem_items = smem_col_cnt * (smem_iter + 1) < items_cnt ? smem_col_cnt : items_cnt - smem_col_cnt * smem_iter;

			// First smem_items threads read one column each, other threads are busy
			// Should be enough threads to read smem_items columns
			if(threadIdx.x < smem_items) {
				int global_col = csr_col_idxs[start + smem_iter * smem_col_cnt + threadIdx.x];
				for(int row = 0; row < f; ++row) {
					smem[f * threadIdx.x + row] = VT[f * global_col + row];
				}
			}

			__syncthreads();

			// actual work

			int left_row = 0;
			int out_row = left_row;

			while(left_row < f) {
				float out = 0;

				for(int item_nr = 0; item_nr < smem_items; ++item_nr) {
					// VT is column-major:
					//     for left matrix, move to next row <=> add 1 to idx
					//     for left matrix, move to next col <=> add f to idx
					//     for top matrix, move to next row <=> add f to idx
					//     for top matrix, move to next col <=> add 1 to idx

					int left_col = item_nr;
					int top_row = left_col;

					int left_row_offset = left_row;
					int left_col_offset = left_col * f;

					int top_row_offset = top_row * f;
					int top_col_offset = top_col;

					out += smem[left_row_offset + left_col_offset] * smem[top_row_offset + top_col_offset];
				}

				vtvs[user_idx * f * f + out_row + out_col * f] += out;

				++left_row;
				++out_row;
			}
		}

		// regularization
		vtvs[user_idx * f * f + out_col + out_col * f] += items_cnt * lambda;
	}
}

__global__
void calculate_vtvs_smem_col_major_two_threads(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
	extern __shared__ float smem [];

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int smem_iters = (items_cnt - 1) / smem_col_cnt + 1;


		int top_col = threadIdx.x;
		int out_col = top_col;

		for(int smem_iter = 0; smem_iter < smem_iters; ++smem_iter) {

			int smem_items = smem_col_cnt * (smem_iter + 1) < items_cnt ? smem_col_cnt : items_cnt - smem_col_cnt * smem_iter;

			// First smem_items * 2 threads read one half column each, other threads are busy.
			// Should be enough threads to read smem_items columns
			if(threadIdx.x < smem_items * 2) {
				int global_col = csr_col_idxs[start + smem_iter * smem_col_cnt + threadIdx.x];
				int first_row = f / 2 * threadIdx.x % 2;
				for(int row = first_row; row < f / 2; ++row) {
					smem[f * threadIdx.x / 2 + row] = VT[f * global_col + row];
				}
			}

			__syncthreads();

			// actual work

			int left_row = 0;
			int out_row = left_row;

			while(left_row < f) {
				float out = 0;

				for(int item_nr = 0; item_nr < smem_items; ++item_nr) {
					// VT is column-major:
					//     for left matrix, move to next row <=> add 1 to idx
					//     for left matrix, move to next col <=> add f to idx
					//     for top matrix, move to next row <=> add f to idx
					//     for top matrix, move to next col <=> add 1 to idx

					int left_col = item_nr;
					int top_row = left_col;

					int left_row_offset = left_row;
					int left_col_offset = left_col * f;

					int top_row_offset = top_row * f;
					int top_col_offset = top_col;

					out += smem[left_row_offset + left_col_offset] * smem[top_row_offset + top_col_offset];
				}

				vtvs[user_idx * f * f + out_row + out_col * f] += out;	// how bad is += for performance?

				++left_row;
				++out_row;
			}
		}

		// regularization
		vtvs[user_idx * f * f + out_col + out_col * f] += items_cnt * lambda;
	}
}

std::string als_model::to_string(CALCULATE_VVTS_TYPE calculate_vvts_type) {
	switch(calculate_vvts_type) {
		case CALCULATE_VVTS_TYPE::SIMPLE: return "SIMPLE";
		case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR: return "SMEM_ROW_MAJOR";
		case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR: return "SMEM_COL_MAJOR";
		case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR_NO_CALC: return "SMEM_ROW_MAJOR_NO_CALC";
		case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR_TWO_THREADS: return "SMEM_COL_MAJOR_TWO_THREADS";
		default: return "UNKNOWN";
	}
}

als_model::als_model(cuda_sparse_matrix &train_ratings, cuda_sparse_matrix &test_ratings, int f, float lambda, int iters, CALCULATE_VVTS_TYPE calculate_vvts_type, int smem_col_cnt):
		train_ratings(train_ratings), test_ratings(test_ratings), f(f), lambda(lambda), iters(iters), calculate_vvts_type(calculate_vvts_type),
		smem_col_cnt(smem_col_cnt) {
	m = train_ratings.row_cnt;
	n = train_ratings.col_cnt;

	CUDA_CHECK(hipHostMalloc((void **)&h_VT, n * f * sizeof(h_VT[0])));
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_VT, n * f * sizeof(d_VT[0])));

	CUDA_CHECK(hipHostMalloc((void **)&h_UT, m * f * sizeof(h_UT[0])));
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_UT, m * f * sizeof(d_UT[0])));

	// на первом этапе без X_BATCH размер f * f * m, затем будет f * f * batch_size
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_vtvs, f * f * m * sizeof(d_vtvs[0])));

	// на первом этапе без THETA_BATCH размер f * f * n, затем будет f * f * batch_size
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_utus, f * f * n * sizeof(d_utus[0])));

	// float *d_VTRT;	// device transposed global item factor matrix multiplied by transposed ratings, f x m (confusing name ythetaT, IMHO thetaTyT is clearer)

	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_VTRT, f * m * sizeof(d_VTRT[0])));

	// float *d_UTR;	// device transposed global user factor matrix multiplied by ratings, f x n (confusing name yTXT, IMHO XTy is clearer)

	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_UTR, f * n * sizeof(d_UTR[0])));

	CUSPARSE_CHECK(hipsparseCreate(&cusparse_handle));
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));
}

als_model::~als_model() {
	CUDA_CHECK(hipHostFree(h_VT));
	CUDA_CHECK(hipFree(d_VT));

	CUDA_CHECK(hipHostFree(h_UT));
	CUDA_CHECK(hipFree(d_UT));

	CUDA_CHECK(hipFree(d_vtvs));
	CUDA_CHECK(hipFree(d_utus));

	CUSPARSE_CHECK(hipsparseDestroy(cusparse_handle));
	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
}

void als_model::train() {

#ifdef USE_LOGGER
	g_logger.log("als model training started", true);
#endif

	unsigned int seed = 0;
	srand (seed);
	for (size_t k = 0; k < n * f; k++)
		h_VT[k] = 0.2*((float)rand() / (float)RAND_MAX);
	for (size_t k = 0; k < m * f; k++)
		h_UT[k] = 0;//0.1*((float) rand() / (float)RAND_MAX);

	CUDA_CHECK(hipMemcpy(d_VT, h_VT, n * f * sizeof(h_VT[0]), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_UT, h_UT, m * f * sizeof(h_UT[0]), hipMemcpyHostToDevice));

#ifdef USE_LOGGER
	g_logger.log("factors initialization done", true);
#endif

	for(size_t it = 0; it < iters; ++it) {

#ifdef USE_LOGGER
		g_logger.als_iter = it + 1;
#endif

		// ---------- update U ----------
		{

#ifdef USE_LOGGER
			g_logger.log("update U started", true);
#endif

			// device ratings multiplied by global item factor matrix, m x f (ytheta)
			float *d_RV;

			// TODO: single array of max(m, n) * f allocated in model constructor

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_RV, m * f * sizeof(d_RV[0])));

			const float alpha = 1.0f;
			const float beta = 0.0f;

			// R * VT.T = RV

			CUSPARSE_CHECK(hipsparseScsrmm2(
					cusparse_handle,
					HIPSPARSE_OPERATION_NON_TRANSPOSE,
					HIPSPARSE_OPERATION_TRANSPOSE,
					m,
					f,
					n,
					train_ratings.val_cnt,
					&alpha,
					train_ratings.cusparse_descr,
					train_ratings.d_csr_coo_vals,
					train_ratings.d_csr_row_ptrs,
					train_ratings.d_csr_coo_col_idxs,
					d_VT,
					f,
					&beta,
					d_RV,
					m
			));

			// (RV).T = VTRT

			CUBLAS_CHECK(hipblasSgeam(
					cublas_handle,
					HIPBLAS_OP_T,
					HIPBLAS_OP_N,
					f,
					m,
					&alpha,
					d_RV,
					m,
					&beta,
					d_VTRT,
					f,
					d_VTRT,
					f
			));

			CUDA_CHECK(hipFree(d_RV));

#ifdef USE_LOGGER
			g_logger.log("VTRT via cuSPARSE and cuBLAS done", true);
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs calculation started type=" + to_string(calculate_vvts_type), true);
#endif

			int smem_size = 0;

			switch (calculate_vvts_type) {
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR:
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR:
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR_NO_CALC:
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR_TWO_THREADS:
				smem_size = smem_col_cnt * f * sizeof(d_VT[0]);
				g_logger.log("vtvs smem_col_cnt=" + std::to_string(smem_col_cnt) + " smem_size=" + std::to_string(smem_size), true);
			}

			switch (calculate_vvts_type) {
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR:
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_row_major<<<m, f, smem_size>>>(d_vtvs, train_ratings.d_csr_row_ptrs,
						train_ratings.d_csr_coo_col_idxs, lambda, m, f, d_VT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR_NO_CALC:
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_row_major_no_calc<<<m, f, smem_size>>>(d_vtvs, train_ratings.d_csr_row_ptrs,
						train_ratings.d_csr_coo_col_idxs, lambda, m, f, d_VT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR:
				if(f < smem_col_cnt) {
					throw std::runtime_error("SMEM_COL_MAJOR: f(" + std::to_string(f) + ") should be greater than or equal to smem_col_cnt("
							+ std::to_string(smem_col_cnt) + ")"
					);
				}
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_col_major<<<m, f, smem_size>>>(d_vtvs, train_ratings.d_csr_row_ptrs,
						train_ratings.d_csr_coo_col_idxs, lambda, m, f, d_VT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR_TWO_THREADS:
				if(f < smem_col_cnt * 2) {
					throw std::runtime_error("SMEM_COL_MAJOR_TWO_THREADS: f(" + std::to_string(f) + ") should be greater than or equal to smem_col_cnt * 2 ("
							+ std::to_string(smem_col_cnt) + " * 2 = " + std::to_string(smem_col_cnt * 2) + ")"
					);
				}
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_col_major_two_threads<<<m, f, smem_size>>>(d_vtvs, train_ratings.d_csr_row_ptrs,
						train_ratings.d_csr_coo_col_idxs, lambda, m, f, d_VT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SIMPLE:
			default:
				// void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
				calculate_vtvs<<<m, f>>>(d_vtvs, train_ratings.d_csr_row_ptrs,
						train_ratings.d_csr_coo_col_idxs, lambda, m, f, d_VT
				);
			}
			CUDA_CHECK(hipPeekAtLastError());

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs calculation done type=" + to_string(calculate_vvts_type), true);
#endif

			// TODO: single array of max(m, n) allocated in model constructor

			// host array of pointers to each device vtv
			float **h_d_vtvs_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_vtvs_ptrs, m * sizeof(h_d_vtvs_ptrs[0])));

			for(size_t i = 0; i < m; ++i) {
				h_d_vtvs_ptrs[i] = &d_vtvs[i * f * f];
			}

			// device array of pointers to each device vtv
			float **d_d_vtvs_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_vtvs_ptrs, m * sizeof(d_d_vtvs_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_vtvs_ptrs, h_d_vtvs_ptrs, m * sizeof(h_d_vtvs_ptrs[0]), hipMemcpyHostToDevice));

			// required by hipblasSgetrfBatched but not used for now
			int *d_getrf_infos;

			CUDA_CHECK(CUDA_MALLOC_DEVICE(&d_getrf_infos, m * sizeof(d_getrf_infos[0])));

			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrfBatched(cublas_handle, f, d_d_vtvs_ptrs, f, NULL, d_getrf_infos, m));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs batched LU factorization done", true);
#endif

			int getrs_info;

			// host array of pointers to each device VTRT column
			float **h_d_VTRT_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_VTRT_ptrs, m * sizeof(h_d_VTRT_ptrs[0])));

			for(size_t i = 0; i < m; ++i) {
				h_d_VTRT_ptrs[i] = &d_VTRT[i * f];
			}

			// device array of pointers to each device VTRT column
			float **d_d_VTRT_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_VTRT_ptrs, m * sizeof(d_d_VTRT_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_VTRT_ptrs, h_d_VTRT_ptrs, m * sizeof(h_d_VTRT_ptrs[0]), hipMemcpyHostToDevice));

			// d_VTRT gets overwritten by result (VT)
			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrsBatched(
					cublas_handle,
					HIPBLAS_OP_N,
					f,
					1,
					(const float * const *)d_d_vtvs_ptrs,
					f,
					nullptr,
					(float * const *)d_d_VTRT_ptrs,
					f,
					&getrs_info,
					m
			));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("U batched solve done", true);
#endif

			// write result
			CUDA_CHECK(hipMemcpy(d_UT, d_VTRT, m * f * sizeof(d_VTRT[0]), hipMemcpyDeviceToDevice));

			CUDA_CHECK(hipHostFree(h_d_vtvs_ptrs));
			CUDA_CHECK(hipFree(d_d_vtvs_ptrs));
			CUDA_CHECK(hipFree(d_getrf_infos));
			CUDA_CHECK(hipHostFree(h_d_VTRT_ptrs));
			CUDA_CHECK(hipFree(d_d_VTRT_ptrs));

#ifdef USE_LOGGER
			g_logger.log("update U done", true);
#endif

		}	// update U block
		// ---------- update V ----------
		{

#ifdef USE_LOGGER
			g_logger.log("update V started", true);
#endif

			// device transposed ratings multiplied by global user factor matrix, m x f (yTX)
			float *d_RTU;

			// TODO: single array of max(m, n) * f allocated in model constructor

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_RTU, n * f * sizeof(d_RTU[0])));

			const float alpha = 1.0f;
			const float beta = 0.0f;

			// RT * UT.T = RTU

			// https://docs.nvidia.com/cuda/cusparse/index.html#csc-format
			// Note: The matrix A in CSR format has exactly the same memory layout as its transpose in CSC format (and vice versa).
			CUSPARSE_CHECK(hipsparseScsrmm2(
					cusparse_handle,
					HIPSPARSE_OPERATION_NON_TRANSPOSE,
					HIPSPARSE_OPERATION_TRANSPOSE,
					n,
					f,
					m,
					train_ratings.val_cnt,
					&alpha,
					train_ratings.cusparse_descr,
					train_ratings.d_csc_vals,
					train_ratings.d_csc_col_ptrs,
					train_ratings.d_csc_row_idxs,
					d_UT,
					f,
					&beta,
					d_RTU,
					n
			));

			// (RTU).T = UTR

			CUBLAS_CHECK(hipblasSgeam(
					cublas_handle,
					HIPBLAS_OP_T,
					HIPBLAS_OP_N,
					f,
					n,
					&alpha,
					d_RTU,
					n,
					&beta,
					d_UTR,
					f,
					d_UTR,
					f
			));

			CUDA_CHECK(hipFree(d_RTU));

#ifdef USE_LOGGER
			g_logger.log("d_UTR via cuSPARSE and cuBLAS done", true);
#endif

#ifdef USE_LOGGER
			g_logger.log("utus calculation started type=" + to_string(calculate_vvts_type), true);
#endif

			// Function is named calculate_vtvs but here we actually calculate utus.
			// Naming is kept for U update for easier debugging

			int smem_size = 0;

			switch (calculate_vvts_type) {
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR:
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR:
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR_NO_CALC:
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR_TWO_THREADS:
				smem_size = smem_col_cnt * f * sizeof(d_UT[0]);
				g_logger.log("utus smem_col_cnt=" + std::to_string(smem_col_cnt) + " smem_size=" + std::to_string(smem_size), true);
			}

			switch (calculate_vvts_type) {
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR:
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_row_major<<<n, f, smem_size>>>(d_utus, train_ratings.d_csc_col_ptrs,
						train_ratings.d_csc_row_idxs, lambda, n, f, d_UT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_ROW_MAJOR_NO_CALC:
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_row_major_no_calc<<<n, f, smem_size>>>(d_utus, train_ratings.d_csc_col_ptrs,
						train_ratings.d_csc_row_idxs, lambda, n, f, d_UT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR:
				if(f < smem_col_cnt) {
					throw std::runtime_error("SMEM_COL_MAJOR: f(" + std::to_string(f) + ") should be greater than or equal to smem_col_cnt("
							+ std::to_string(smem_col_cnt) + ")"
					);
				}
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_col_major<<<n, f, smem_size>>>(d_utus, train_ratings.d_csc_col_ptrs,
						train_ratings.d_csc_row_idxs, lambda, n, f, d_UT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SMEM_COL_MAJOR_TWO_THREADS:
				if(f < smem_col_cnt * 2) {
					throw std::runtime_error("SMEM_COL_MAJOR_TWO_THREADS: f(" + std::to_string(f) + ") should be greater than or equal to smem_col_cnt * 2 ("
							+ std::to_string(smem_col_cnt) + " * 2 = " + std::to_string(smem_col_cnt * 2) + ")"
					);
				}
				// void calculate_vtvs_smem_row_major(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT, int smem_col_cnt) {
				calculate_vtvs_smem_col_major_two_threads<<<n, f, smem_size>>>(d_utus, train_ratings.d_csc_col_ptrs,
						train_ratings.d_csc_row_idxs, lambda, n, f, d_UT, smem_col_cnt
				);
				break;
			case CALCULATE_VVTS_TYPE::SIMPLE:
			default:
				// void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
				calculate_vtvs<<<n, f>>>(d_utus, train_ratings.d_csc_col_ptrs,
						train_ratings.d_csc_row_idxs, lambda, n, f, d_UT
				);
			}

			CUDA_CHECK(hipPeekAtLastError());

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("utus calculation done", true);
#endif

			// TODO: single array of max(m, n) allocated in model constructor

			// host array of pointers to each device utu
			float **h_d_utus_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_utus_ptrs, n * sizeof(h_d_utus_ptrs[0])));

			for(size_t i = 0; i < n; ++i) {
				h_d_utus_ptrs[i] = &d_utus[i * f * f];
			}

			// device array of pointers to each device utu
			float **d_d_utus_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_utus_ptrs, n * sizeof(d_d_utus_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_utus_ptrs, h_d_utus_ptrs, n * sizeof(h_d_utus_ptrs[0]), hipMemcpyHostToDevice));

			// required by hipblasSgetrfBatched but not used for now
			int *d_getrf_infos;

			CUDA_CHECK(CUDA_MALLOC_DEVICE(&d_getrf_infos, n * sizeof(d_getrf_infos[0])));

			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrfBatched(cublas_handle, f, d_d_utus_ptrs, f, NULL, d_getrf_infos, n));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("utus batched LU factorization done type=" + to_string(calculate_vvts_type), true);
#endif

			int getrs_info;

			// host array of pointers to each device UTR column
			float **h_d_UTR_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_UTR_ptrs, n * sizeof(h_d_UTR_ptrs[0])));

			for(size_t i = 0; i < n; ++i) {
				h_d_UTR_ptrs[i] = &d_UTR[i * f];
			}

			// device array of pointers to each device UTR column
			float **d_d_UTR_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_UTR_ptrs, n * sizeof(d_d_utus_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_UTR_ptrs, h_d_UTR_ptrs, n * sizeof(h_d_UTR_ptrs[0]), hipMemcpyHostToDevice));

			// d_UTR gets overwritten by result (UT)
			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrsBatched(
					cublas_handle,
					HIPBLAS_OP_N,
					f,
					1,
					(const float * const *)d_d_utus_ptrs,
					f,
					nullptr,
					(float * const *)d_d_UTR_ptrs,
					f,
					&getrs_info,
					n
			));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("V batched solve done", true);
#endif

			// write result
			CUDA_CHECK(hipMemcpy(d_VT, d_UTR, n * f * sizeof(d_UTR[0]), hipMemcpyDeviceToDevice));

			CUDA_CHECK(hipHostFree(h_d_utus_ptrs));
			CUDA_CHECK(hipFree(d_d_utus_ptrs));
			CUDA_CHECK(hipFree(d_getrf_infos));
			CUDA_CHECK(hipHostFree(h_d_UTR_ptrs));
			CUDA_CHECK(hipFree(d_d_UTR_ptrs));

#ifdef USE_LOGGER
			g_logger.log("update V done", true);
#endif

		}	// update V block
	}	// iters loop

#ifdef USE_LOGGER
	g_logger.als_iter = 0;
#endif

	// final result from device to host

	CUDA_CHECK(hipMemcpy(h_VT, d_VT, m * f, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_UT, d_UT, n * f, hipMemcpyDeviceToHost));

#ifdef USE_LOGGER
	g_logger.log("als model training done", true);
#endif
}


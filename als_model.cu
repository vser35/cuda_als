#include "hip/hip_runtime.h"
#include <memory>


#include "als_model.h"
#include "hip/hip_runtime.h"
#include "cuda_common.h"
#include "logger.h"

/*
auto cuda_malloc_device = [](size_t size) {
	void *ptr;
	CUDA_CHECK(CUDA_MALLOC_DEVICE(&ptr, size));
	return ptr;
};

auto cuda_deleter_device = [](void *ptr) {
	CUDA_CHECK(hipFree(ptr));
};
*/

__global__
void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
	// start = current row pointer
	// end = next row pointer
	// nr_of_cols = end - start
	// thread_col = threadIdx.x
	// for i = 0 to f (from up to down)
	//     for j = 0 to nr_of_cols
	//         curr_col_in_global = csr_col_idxs[start + j]
	//         tmp += VT[curr_col_in_global * f + i] * VT[curr_col_in_global * f + i]
	//     vtvs[blockIdx.x * f * f + i] = tmp
	//

	// left matrix is vt (i.e. columns from VT for items rated by current user)
	// top matrix is v (i.e. vt transposed)

	/* start = current row pointer
	 * end = next row pointer
	 * nr_of_items = end - start
	 * top_col = out_col = threadIdx.x
	 * for(left_row = out_row = 0; left_row < f; ++left_row ++ out_row)
	 *     temp = 0
	 *     for(curr_item_nr = 0; curr_item_nr < nr_of_items; ++curr_item)
	 *         left_col = top_row = curr_item_nr
	 *         // VT is column-major:
	 *         //     for left matrix, move to next row <=> add 1 to idx
	 *         //     for left matrix, move to next col <=> add f to idx
	 *         //     for top matrix, move to next row <=> add f to idx
	 *         //     for top matrix, move to next col <=> add 1 to idx
	 *
	 *         left_row_offset = left_row
	 *         left_col_offset = csr_col_idxs[start + left_col] * f
	 *
	 *         top_row_offset = csr_col_idxs[start + top_row] * f
	 *         top_col_offset = top_row
	 *
	 *         temp += VT[left_row_offset + left_col_offset] * VT[top_row_offset + top_col_offset]
	 *     vtvs[blockIdx.x * f * f + left_row + ] = temp
	 */

	if(threadIdx.x < f) {
		int user_idx = blockIdx.x;

		int start = csr_row_ptrs[user_idx];
		int end = csr_row_ptrs[user_idx + 1];

		int items_cnt = end - start;

		int top_col = threadIdx.x;
		int out_col = top_col;

		int left_row = 0;
		int out_row = left_row;

		while(left_row < f) {
			float out = 0;

			for(int item_nr = 0; item_nr < items_cnt; ++ item_nr) {
				int left_col = item_nr;
				int top_row = left_col;

				// VT is column-major:
				//     for left matrix, move to next row <=> add 1 to idx
				//     for left matrix, move to next col <=> add f to idx
				//     for top matrix, move to next row <=> add f to idx
				//     for top matrix, move to next col <=> add 1 to idx

				int left_row_offset = left_row;
				int left_col_offset = csr_col_idxs[start + left_col] * f;

				int top_row_offset = csr_col_idxs[start + top_row] * f;
				int top_col_offset = top_col;

				out += VT[left_row_offset + left_col_offset] * VT[top_row_offset + top_col_offset];
			}

			// regularization
			if(left_row == top_col) {
				out += items_cnt * lambda;
			}

			vtvs[user_idx * f * f + out_row + out_col * f] = out;

			++left_row;
			++out_row;
		}
	}
}

als_model::als_model(cuda_sparse_matrix &train_ratings, cuda_sparse_matrix &test_ratings, int f, float lambda, int iters):
		train_ratings(train_ratings), test_ratings(test_ratings), f(f), lambda(lambda), iters(iters) {
	m = train_ratings.row_cnt;
	n = train_ratings.col_cnt;

	CUDA_CHECK(hipHostMalloc((void **)&h_VT, n * f * sizeof(h_VT[0])));
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_VT, n * f * sizeof(d_VT[0])));

	CUDA_CHECK(hipHostMalloc((void **)&h_UT, m * f * sizeof(h_UT[0])));
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_UT, m * f * sizeof(d_UT[0])));

	// на первом этапе без X_BATCH размер f * f * m, затем будет f * f * batch_size
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_vtvs, f * f * m * sizeof(d_vtvs[0])));

	// на первом этапе без THETA_BATCH размер f * f * n, затем будет f * f * batch_size
	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_utus, f * f * n * sizeof(d_utus[0])));

	// float *d_VTRT;	// device transposed global item factor matrix multiplied by transposed ratings, f x m (confusing name ythetaT, IMHO thetaTyT is clearer)

	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_VTRT, f * m * sizeof(d_VTRT[0])));

	// float *d_UTR;	// device transposed global user factor matrix multiplied by ratings, f x n (confusing name yTXT, IMHO XTy is clearer)

	CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_UTR, f * n * sizeof(d_UTR[0])));

	CUSPARSE_CHECK(hipsparseCreate(&cusparse_handle));
	CUBLAS_CHECK(hipblasCreate(&cublas_handle));
}

als_model::~als_model() {
	CUDA_CHECK(hipHostFree(h_VT));
	CUDA_CHECK(hipFree(d_VT));

	CUDA_CHECK(hipHostFree(h_UT));
	CUDA_CHECK(hipFree(d_UT));

	CUDA_CHECK(hipFree(d_vtvs));
	CUDA_CHECK(hipFree(d_utus));

	CUSPARSE_CHECK(hipsparseDestroy(cusparse_handle));
	CUBLAS_CHECK(hipblasDestroy(cublas_handle));
}

void als_model::train() {

#ifdef USE_LOGGER
	g_logger.log("als model training started", true);
#endif

	unsigned int seed = 0;
	srand (seed);
	for (size_t k = 0; k < n * f; k++)
		h_VT[k] = 0.2*((float)rand() / (float)RAND_MAX);
	for (size_t k = 0; k < m * f; k++)
		h_UT[k] = 0;//0.1*((float) rand() / (float)RAND_MAX);

	CUDA_CHECK(hipMemcpy(d_VT, h_VT, n * f * sizeof(h_VT[0]), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_UT, h_UT, m * f * sizeof(h_UT[0]), hipMemcpyHostToDevice));

#ifdef USE_LOGGER
	g_logger.log("factors initialization done", true);
#endif

	for(size_t it = 0; it < iters; ++it) {

#ifdef USE_LOGGER
		g_logger.als_iter = it + 1;
#endif

		// ---------- update U ----------
		{

#ifdef USE_LOGGER
			g_logger.log("update U started", true);
#endif

			// device ratings multiplied by global item factor matrix, m x f (ytheta)
			float *d_RV;

			// TODO: single array of max(m, n) * f allocated in model constructor

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_RV, m * f * sizeof(d_RV[0])));

			const float alpha = 1.0f;
			const float beta = 0.0f;

			// R * VT.T = RV

			CUSPARSE_CHECK(hipsparseScsrmm2(
					cusparse_handle,
					HIPSPARSE_OPERATION_NON_TRANSPOSE,
					HIPSPARSE_OPERATION_TRANSPOSE,
					m,
					f,
					n,
					train_ratings.val_cnt,
					&alpha,
					train_ratings.cusparse_descr,
					train_ratings.d_csr_coo_vals,
					train_ratings.d_csr_row_ptrs,
					train_ratings.d_csr_coo_col_idxs,
					d_VT,
					f,
					&beta,
					d_RV,
					m
			));

			// (RV).T = VTRT

			CUBLAS_CHECK(hipblasSgeam(
					cublas_handle,
					HIPBLAS_OP_T,
					HIPBLAS_OP_N,
					f,
					m,
					&alpha,
					d_RV,
					m,
					&beta,
					d_VTRT,
					f,
					d_VTRT,
					f
			));

			CUDA_CHECK(hipFree(d_RV));

#ifdef USE_LOGGER
			g_logger.log("VTRT via cuSPARSE and cuBLAS done", true);
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs calculation started", true);
#endif

			// void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
			calculate_vtvs<<<m, f>>>(
					d_vtvs,
					train_ratings.d_csr_row_ptrs,
					train_ratings.d_csr_coo_col_idxs,
					lambda,
					m,
					f,
					d_VT
			);

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs calculation done", true);
#endif

			// TODO: single array of max(m, n) allocated in model constructor

			// host array of pointers to each device vtv
			float **h_d_vtvs_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_vtvs_ptrs, m * sizeof(h_d_vtvs_ptrs[0])));

			for(size_t i = 0; i < m; ++i) {
				h_d_vtvs_ptrs[i] = &d_vtvs[i * f * f];
			}

			// device array of pointers to each device vtv
			float **d_d_vtvs_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_vtvs_ptrs, m * sizeof(d_d_vtvs_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_vtvs_ptrs, h_d_vtvs_ptrs, m * sizeof(h_d_vtvs_ptrs[0]), hipMemcpyHostToDevice));

			// required by hipblasSgetrfBatched but not used for now
			int *d_getrf_infos;

			CUDA_CHECK(CUDA_MALLOC_DEVICE(&d_getrf_infos, m * sizeof(d_getrf_infos[0])));

			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrfBatched(cublas_handle, f, d_d_vtvs_ptrs, f, NULL, d_getrf_infos, m));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("vtvs batched LU factorization done", true);
#endif

			int getrs_info;

			// host array of pointers to each device VTRT column
			float **h_d_VTRT_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_VTRT_ptrs, m * sizeof(h_d_VTRT_ptrs[0])));

			for(size_t i = 0; i < m; ++i) {
				h_d_VTRT_ptrs[i] = &d_VTRT[i * f];
			}

			// device array of pointers to each device VTRT column
			float **d_d_VTRT_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_VTRT_ptrs, m * sizeof(d_d_VTRT_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_VTRT_ptrs, h_d_VTRT_ptrs, m * sizeof(h_d_VTRT_ptrs[0]), hipMemcpyHostToDevice));

			// d_VTRT gets overwritten by result (VT)
			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrsBatched(
					cublas_handle,
					HIPBLAS_OP_N,
					f,
					1,
					(const float * const *)d_d_vtvs_ptrs,
					f,
					nullptr,
					(float * const *)d_d_VTRT_ptrs,
					f,
					&getrs_info,
					m
			));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("U batched solve done", true);
#endif

			// write result
			CUDA_CHECK(hipMemcpy(d_UT, d_VTRT, m * f * sizeof(d_VTRT[0]), hipMemcpyDeviceToDevice));

			CUDA_CHECK(hipHostFree(h_d_vtvs_ptrs));
			CUDA_CHECK(hipFree(d_d_vtvs_ptrs));
			CUDA_CHECK(hipFree(d_getrf_infos));
			CUDA_CHECK(hipHostFree(h_d_VTRT_ptrs));
			CUDA_CHECK(hipFree(d_d_VTRT_ptrs));

#ifdef USE_LOGGER
			g_logger.log("update U done", true);
#endif

		}	// update U block
		// ---------- update V ----------
		{

#ifdef USE_LOGGER
			g_logger.log("update V started", true);
#endif

			// device transposed ratings multiplied by global user factor matrix, m x f (yTX)
			float *d_RTU;

			// TODO: single array of max(m, n) * f allocated in model constructor

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_RTU, n * f * sizeof(d_RTU[0])));

			const float alpha = 1.0f;
			const float beta = 0.0f;

			// RT * UT.T = RTU

			// https://docs.nvidia.com/cuda/cusparse/index.html#csc-format
			// Note: The matrix A in CSR format has exactly the same memory layout as its transpose in CSC format (and vice versa).
			CUSPARSE_CHECK(hipsparseScsrmm2(
					cusparse_handle,
					HIPSPARSE_OPERATION_NON_TRANSPOSE,
					HIPSPARSE_OPERATION_TRANSPOSE,
					n,
					f,
					m,
					train_ratings.val_cnt,
					&alpha,
					train_ratings.cusparse_descr,
					train_ratings.d_csc_vals,
					train_ratings.d_csc_col_ptrs,
					train_ratings.d_csc_row_idxs,
					d_UT,
					f,
					&beta,
					d_RTU,
					n
			));

			// (RTU).T = UTR

			CUBLAS_CHECK(hipblasSgeam(
					cublas_handle,
					HIPBLAS_OP_T,
					HIPBLAS_OP_N,
					f,
					n,
					&alpha,
					d_RTU,
					n,
					&beta,
					d_UTR,
					f,
					d_UTR,
					f
			));

			CUDA_CHECK(hipFree(d_RTU));

#ifdef USE_LOGGER
			g_logger.log("d_UTR via cuSPARSE and cuBLAS done", true);
#endif

#ifdef USE_LOGGER
			g_logger.log("utus calculation started", true);
#endif

			// Function is named calculate_vtvs but here we actually calculate utus.
			// Naming is kept for U update for easier debugging
			// void calculate_vtvs(float *vtvs, int *csr_row_ptrs, int *csr_col_idxs, float lambda, int m, int f, float *VT) {
			calculate_vtvs<<<n, f>>>(
					d_utus,
					train_ratings.d_csc_col_ptrs,
					train_ratings.d_csc_row_idxs,
					lambda,
					n,
					f,
					d_UT
			);

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("utus calculation done", true);
#endif

			// TODO: single array of max(m, n) allocated in model constructor

			// host array of pointers to each device utu
			float **h_d_utus_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_utus_ptrs, n * sizeof(h_d_utus_ptrs[0])));

			for(size_t i = 0; i < n; ++i) {
				h_d_utus_ptrs[i] = &d_utus[i * f * f];
			}

			// device array of pointers to each device utu
			float **d_d_utus_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_utus_ptrs, n * sizeof(d_d_utus_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_utus_ptrs, h_d_utus_ptrs, n * sizeof(h_d_utus_ptrs[0]), hipMemcpyHostToDevice));

			// required by hipblasSgetrfBatched but not used for now
			int *d_getrf_infos;

			CUDA_CHECK(CUDA_MALLOC_DEVICE(&d_getrf_infos, n * sizeof(d_getrf_infos[0])));

			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrfBatched(cublas_handle, f, d_d_utus_ptrs, f, NULL, d_getrf_infos, n));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("utus batched LU factorization done", true);
#endif

			int getrs_info;

			// host array of pointers to each device UTR column
			float **h_d_UTR_ptrs;

			CUDA_CHECK(hipHostMalloc((void **)&h_d_UTR_ptrs, n * sizeof(h_d_UTR_ptrs[0])));

			for(size_t i = 0; i < n; ++i) {
				h_d_UTR_ptrs[i] = &d_UTR[i * f];
			}

			// device array of pointers to each device UTR column
			float **d_d_UTR_ptrs;

			CUDA_CHECK(CUDA_MALLOC_DEVICE((void **)&d_d_UTR_ptrs, n * sizeof(d_d_utus_ptrs[0])));
			CUDA_CHECK(hipMemcpy(d_d_UTR_ptrs, h_d_UTR_ptrs, n * sizeof(h_d_UTR_ptrs[0]), hipMemcpyHostToDevice));

			// d_UTR gets overwritten by result (UT)
			// stepping here in Nsight debug session causes GDB crash so don't put breakpoints here
			CUBLAS_CHECK(hipblasSgetrsBatched(
					cublas_handle,
					HIPBLAS_OP_N,
					f,
					1,
					(const float * const *)d_d_utus_ptrs,
					f,
					nullptr,
					(float * const *)d_d_UTR_ptrs,
					f,
					&getrs_info,
					n
			));

#if defined (DEBUG) || defined(USE_LOGGER)
			CUDA_CHECK(hipDeviceSynchronize());
#endif

#ifdef USE_LOGGER
			g_logger.log("V batched solve done", true);
#endif

			// write result
			CUDA_CHECK(hipMemcpy(d_VT, d_UTR, n * f * sizeof(d_UTR[0]), hipMemcpyDeviceToDevice));

			CUDA_CHECK(hipHostFree(h_d_utus_ptrs));
			CUDA_CHECK(hipFree(d_d_utus_ptrs));
			CUDA_CHECK(hipFree(d_getrf_infos));
			CUDA_CHECK(hipHostFree(h_d_UTR_ptrs));
			CUDA_CHECK(hipFree(d_d_UTR_ptrs));

#ifdef USE_LOGGER
			g_logger.log("update V done", true);
#endif

		}	// update V block
	}	// iters loop

#ifdef USE_LOGGER
	g_logger.als_iter = 0;
#endif

	// final result from device to host

	CUDA_CHECK(hipMemcpy(h_VT, d_VT, m * f, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_UT, d_UT, n * f, hipMemcpyDeviceToHost));

#ifdef USE_LOGGER
	g_logger.log("als model training done", true);
#endif
}

